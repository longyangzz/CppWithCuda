#include "cudaPrint.cuh"
#include "hip/hip_runtime.h"
#include "stdio.h"
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
void CUDA_Printf(int argc, char* argv[]) {
	printf("this is CUDA Printf");

    int devID;
    hipDeviceProp_t deviceProps;
    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char**)argv);

    // get device name
    hipGetDeviceProperties(&deviceProps, devID);
    printf("CUDA device [%s]\n", deviceProps.name);
}